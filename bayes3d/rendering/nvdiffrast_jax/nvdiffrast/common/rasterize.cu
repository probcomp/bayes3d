#include "hip/hip_runtime.h"
// Copyright (c) 2020, NVIDIA CORPORATION.  All rights reserved.
//
// NVIDIA CORPORATION and its licensors retain all intellectual property
// and proprietary rights in and to this software, related documentation
// and any modifications thereto.  Any use, reproduction, disclosure or
// distribution of this software and related documentation without an express
// license agreement from NVIDIA CORPORATION is strictly prohibited.

#include "common.h"
#include "rasterize.h"

//------------------------------------------------------------------------
// Cuda forward rasterizer pixel shader kernel.

__global__ void RasterizeCudaFwdShaderKernel(const RasterizeCudaFwdShaderParams p)
{
    // Calculate pixel position.
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    int pz = blockIdx.z;
    if (px >= p.width || py >= p.height || pz >= p.depth)
        return;

    // Pixel index.
    int pidx = px + p.width * (py + p.height * pz);

    // Fetch triangle idx.
    int triIdx = p.in_idx[pidx] - 1;
    if (triIdx < 0 || triIdx >= p.numTriangles)
    {
        // No or corrupt triangle.
        ((float4*)p.out)[pidx] = make_float4(0.0, 0.0, 0.0, 0.0); // Clear out.
        ((float4*)p.out_db)[pidx] = make_float4(0.0, 0.0, 0.0, 0.0); // Clear out_db.
        return;
    }

    // Fetch vertex indices.
    int vi0 = p.tri[triIdx * 3 + 0];
    int vi1 = p.tri[triIdx * 3 + 1];
    int vi2 = p.tri[triIdx * 3 + 2];

    // Bail out if vertex indices are corrupt.
    if (vi0 < 0 || vi0 >= p.numVertices ||
        vi1 < 0 || vi1 >= p.numVertices ||
        vi2 < 0 || vi2 >= p.numVertices)
        return;

    // In instance mode, adjust vertex indices by minibatch index.
    if (p.instance_mode)
    {
        vi0 += pz * p.numVertices;
        vi1 += pz * p.numVertices;
        vi2 += pz * p.numVertices;
    }

    // Fetch vertex positions.
    float4 p0 = ((float4*)p.pos)[vi0];
    float4 p1 = ((float4*)p.pos)[vi1];
    float4 p2 = ((float4*)p.pos)[vi2];

    // Evaluate edge functions.
    float fx = p.xs * (float)px + p.xo;
    float fy = p.ys * (float)py + p.yo;
    float p0x = p0.x - fx * p0.w;
    float p0y = p0.y - fy * p0.w;
    float p1x = p1.x - fx * p1.w;
    float p1y = p1.y - fy * p1.w;
    float p2x = p2.x - fx * p2.w;
    float p2y = p2.y - fy * p2.w;
    float a0 = p1x*p2y - p1y*p2x;
    float a1 = p2x*p0y - p2y*p0x;
    float a2 = p0x*p1y - p0y*p1x;

    // Perspective correct, normalized barycentrics.
    float iw = 1.f / (a0 + a1 + a2);
    float b0 = a0 * iw;
    float b1 = a1 * iw;

    // Compute z/w for depth buffer.
    float z = p0.z * a0 + p1.z * a1 + p2.z * a2;
    float w = p0.w * a0 + p1.w * a1 + p2.w * a2;
    float zw = z / w;

    // Clamps to avoid NaNs.
    b0 = __saturatef(b0); // Clamp to [+0.0, 1.0].
    b1 = __saturatef(b1); // Clamp to [+0.0, 1.0].
    zw = fmaxf(fminf(zw, 1.f), -1.f);

    // Emit output.
    ((float4*)p.out)[pidx] = make_float4(b0, b1, zw, (float)(triIdx + 1));

    // Calculate bary pixel differentials.
    float dfxdx = p.xs * iw;
    float dfydy = p.ys * iw;
    float da0dx = p2.y*p1.w - p1.y*p2.w;
    float da0dy = p1.x*p2.w - p2.x*p1.w;
    float da1dx = p0.y*p2.w - p2.y*p0.w;
    float da1dy = p2.x*p0.w - p0.x*p2.w;
    float da2dx = p1.y*p0.w - p0.y*p1.w;
    float da2dy = p0.x*p1.w - p1.x*p0.w;
    float datdx = da0dx + da1dx + da2dx;
    float datdy = da0dy + da1dy + da2dy;
    float dudx = dfxdx * (b0 * datdx - da0dx);
    float dudy = dfydy * (b0 * datdy - da0dy);
    float dvdx = dfxdx * (b1 * datdx - da1dx);
    float dvdy = dfydy * (b1 * datdy - da1dy);

    // Emit bary pixel differentials.
    ((float4*)p.out_db)[pidx] = make_float4(dudx, dudy, dvdx, dvdy);
}


__device__ inline void mv_multiply_4(float* matrix, float* vector, float* res)
{
    for (int i = 0; i < 4; i++)
    {
        for (int j = 0; j < 4; j++)
        {
            res[i] += matrix[4*i + j] * vector[j];
        }
    }
}


//------------------------------------------------------------------------
// Gradient Cuda kernel.

template <bool ENABLE_DB>
static __forceinline__ __device__ void RasterizeGradKernelTemplate(const RasterizeGradParams p)
{
    // Temporary space for coalesced atomics.
    CA_DECLARE_TEMP(RAST_GRAD_MAX_KERNEL_BLOCK_WIDTH * RAST_GRAD_MAX_KERNEL_BLOCK_HEIGHT);

    // Calculate pixel position.
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    int pz = blockIdx.z;
    if (px >= p.width || py >= p.height || pz >= p.depth)
        return;

    // Pixel index.
    int pidx = px + p.width * (py + p.height * pz);

    // Read triangle idx and dy.
    float2 dy  = ((float2*)p.dy)[pidx * 2];
    float4 ddb = ENABLE_DB ? ((float4*)p.ddb)[pidx] : make_float4(0.f, 0.f, 0.f, 0.f);

    int triIdx = (int)(((float*)p.out)[pidx * 4 + 3]) - 1;

    int object_idx = (int)(((int*)p.out2)[pidx * 4 + 0]) - 1;
    int depth = p.depth;
    float* pose = p.pose + ( depth * 16 * object_idx +  pz * 16);

    // Exit if nothing to do.
    if (triIdx < 0 || triIdx >= p.numTriangles)
        return; // No or corrupt triangle.
    int grad_all_dy = __float_as_int(dy.x) | __float_as_int(dy.y); // Bitwise OR of all incoming gradients.
    int grad_all_ddb = 0;
    if (ENABLE_DB)
        grad_all_ddb = __float_as_int(ddb.x) | __float_as_int(ddb.y) | __float_as_int(ddb.z) | __float_as_int(ddb.w);
    if (((grad_all_dy | grad_all_ddb) << 1) == 0)
        return; // All incoming gradients are +0/-0.

    // Fetch vertex indices.
    int vi0 = p.tri[triIdx * 3 + 0];
    int vi1 = p.tri[triIdx * 3 + 1];
    int vi2 = p.tri[triIdx * 3 + 2];

    // Bail out if vertex indices are corrupt.
    if (vi0 < 0 || vi0 >= p.numVertices ||
        vi1 < 0 || vi1 >= p.numVertices ||
        vi2 < 0 || vi2 >= p.numVertices)
        return;

    // // In instance mode, adjust vertex indices by minibatch index.
    // if (p.instance_mode)
    // {
    //     vi0 += pz * p.numVertices;
    //     vi1 += pz * p.numVertices;
    //     vi2 += pz * p.numVertices;
    // }

    float* proj = p.proj;

    // Initialize coalesced atomics.
    CA_SET_GROUP(triIdx);

    // Apply projection_matrix  * pose[frame_idx, object_index] * p.pos (which is in object space) to get the clip vertex positions.
    // The following computations assume p.pos is in clip space.
    float* vertex_1_object_frame = p.pos + vi0 * 4;
    float* vertex_2_object_frame = p.pos + vi1 * 4;
    float* vertex_3_object_frame = p.pos + vi2 * 4;

    float vertex_1_camera_frame[4] = {0};
    float vertex_2_camera_frame[4] = {0};
    float vertex_3_camera_frame[4] = {0};
    mv_multiply_4(pose, vertex_1_object_frame, vertex_1_camera_frame);
    mv_multiply_4(pose, vertex_2_object_frame, vertex_2_camera_frame);
    mv_multiply_4(pose, vertex_3_object_frame, vertex_3_camera_frame);

    float vertex_1_clip_space[4] = {0};
    float vertex_2_clip_space[4] = {0};
    float vertex_3_clip_space[4] = {0};
    mv_multiply_4(proj, vertex_1_object_frame, vertex_1_clip_space);
    mv_multiply_4(proj, vertex_2_object_frame, vertex_2_clip_space);
    mv_multiply_4(proj, vertex_3_object_frame, vertex_3_clip_space);

    // Fetch vertex positions.
    // float4 p0 = ((float4*)p.pos)[vi0];
    // float4 p1 = ((float4*)p.pos)[vi1];
    // float4 p2 = ((float4*)p.pos)[vi2];
    float4 p0 = ((float4*)vertex_1_clip_space)[0];
    float4 p1 = ((float4*)vertex_2_clip_space)[0];
    float4 p2 = ((float4*)vertex_3_clip_space)[0];

    // Evaluate edge functions.
    float fx = p.xs * (float)px + p.xo;
    float fy = p.ys * (float)py + p.yo;
    float p0x = p0.x - fx * p0.w;
    float p0y = p0.y - fy * p0.w;
    float p1x = p1.x - fx * p1.w;
    float p1y = p1.y - fy * p1.w;
    float p2x = p2.x - fx * p2.w;
    float p2y = p2.y - fy * p2.w;
    float a0 = p1x*p2y - p1y*p2x;
    float a1 = p2x*p0y - p2y*p0x;
    float a2 = p0x*p1y - p0y*p1x;

    // Compute inverse area with epsilon.
    float at = a0 + a1 + a2;
    float ep = copysignf(1e-6f, at); // ~1 pixel in 1k x 1k image.
    float iw = 1.f / (at + ep);

    // Perspective correct, normalized barycentrics.
    float b0 = a0 * iw;
    float b1 = a1 * iw;

    // Position gradients.
    float gb0  = dy.x * iw;
    float gb1  = dy.y * iw;
    float gbb  = gb0 * b0 + gb1 * b1;
    float gp0x = gbb * (p2y - p1y) - gb1 * p2y;
    float gp1x = gbb * (p0y - p2y) + gb0 * p2y;
    float gp2x = gbb * (p1y - p0y) - gb0 * p1y + gb1 * p0y;
    float gp0y = gbb * (p1x - p2x) + gb1 * p2x;
    float gp1y = gbb * (p2x - p0x) - gb0 * p2x;
    float gp2y = gbb * (p0x - p1x) + gb0 * p1x - gb1 * p0x;
    float gp0w = -fx * gp0x - fy * gp0y;
    float gp1w = -fx * gp1x - fy * gp1y;
    float gp2w = -fx * gp2x - fy * gp2y;

    float loss_grad_v1_clip_space[4] = {gp0x, gp0y, 0.f, gp0w};
    float loss_grad_v2_clip_space[4] = {gp1x, gp1y, 0.f, gp1w};
    float loss_grad_v3_clip_space[4] = {gp2x, gp2y, 0.f, gp2w};

    for (int i = 0; i < 16; i++) {
        int row=i/4, col=i%4;
        //XXX somehow get the xyzw of the vertices
        //col-th coordinate of vi0-, vi1-, vi2-th vertices
        float  vertex_term1 = ((float*) vertex_1_object_frame)[col];
        float  vertex_term2 = ((float*) vertex_2_object_frame)[col];
        float  vertex_term3 = ((float*) vertex_3_object_frame)[col];

        float accumulated_gradient = 0.0;
        accumulated_gradient += loss_grad_v1_clip_space[0] * proj[row] * vertex_term1;
        accumulated_gradient += loss_grad_v2_clip_space[0] * proj[row] * vertex_term2;
        accumulated_gradient += loss_grad_v3_clip_space[0] * proj[row] * vertex_term3;

        accumulated_gradient += loss_grad_v1_clip_space[1] * proj[4 + row] * vertex_term1;
        accumulated_gradient += loss_grad_v2_clip_space[1] * proj[4 + row]* vertex_term2;
        accumulated_gradient += loss_grad_v3_clip_space[1] * proj[4 + row]* vertex_term3;

        accumulated_gradient += loss_grad_v1_clip_space[3] * proj[12 + row] * vertex_term1;
        accumulated_gradient += loss_grad_v2_clip_space[3] * proj[12 + row]* vertex_term2;
        accumulated_gradient += loss_grad_v3_clip_space[3] * proj[12 + row]* vertex_term3;

        // Fix this;
        caAtomicAdd(
            p.grad + (depth * 16 * object_idx +  pz * 16 + i),
            accumulated_gradient
        );
    }
}

// Template specializations.
__global__ void RasterizeGradKernel  (const RasterizeGradParams p) { RasterizeGradKernelTemplate<false>(p); }
__global__ void RasterizeGradKernelDb(const RasterizeGradParams p) { RasterizeGradKernelTemplate<true>(p); }

//------------------------------------------------------------------------
